#include "hip/hip_runtime.h"
#include <stdio.h>
#include <omp.h>
#include <Eigen/Dense>
#include "AffinityMatrix.cuh"
#include "LaplacianMatrix.cuh"
#include "DataProcessing.h"
#include "DS_definitions.h"
#include "DS_timer.h"
#include <utility>
#include <vector>


#include <hipsolver.h>

using namespace Eigen;
using namespace std;

#define TOL 1e-6

void swap(double& x1, double& x2) {
	double temp = x1;
	x1 = x2;
	x2 = temp;
}

void labelDecomposition(MatrixXd& eigenVectors, int* results, int n) {
	std::vector<std::pair<double, int>> dictionary;
	int size = 0;
	for (int i = 0; i < n; i++) {
		if (dictionary.empty()) {
			std::pair<double, int> item(eigenVectors(i, 1), 1);
			dictionary.push_back(item);
			size++;
		}
		else {
			for (int j = 0; j < size; j++) {
				if (abs(dictionary[j].first - eigenVectors(i, 1)) < 1e-3) {
					dictionary[j].second += 1;
					break;
				}
				if (j == size - 1) {
					std::pair<double, int> item(eigenVectors(i, 1), 1);
					dictionary.push_back(item);
					size++;
				}
			}
		}
	}

	for (int i = 0; i < 2; i++) {
		int max = i;
		for (int j = i + 1; j < size; j++) {
			if (dictionary[max].second < dictionary[j].second) {
				max = j;
			}
		}
		std::swap(dictionary[max], dictionary[i]);
	}

	double labels[2];
	labels[0] = dictionary[0].first;
	labels[1] = dictionary[1].first;

	//printf("Size = %d\n", size);
	//printf("label 1 %lf\nlabel 2 %lf\n", labels[0], labels[1]);

	for (int i = 0; i < n; i++) {
		if (abs(eigenVectors(i, 1) - labels[0]) > abs(eigenVectors(i, 1) - labels[1])) {
			results[i] = 1;
		}
		else {
			results[i] = -1;
		}
	}
}

void find_2nd_Min(VectorXd& eigenValue, MatrixXd& eigenVector) {
	int n = eigenVector.rows();

	for (int i = 0; i < 2; i++) {
		int min = i;
		for (int j = 1; j < n; j++) {
			if (eigenValue[min] > eigenValue[j]) {
				min = j;
			}
		}
		swap(eigenValue[min], eigenValue[i]);
		for (int j = 0; j < n; j++) {
			swap(eigenVector(j, min), eigenVector(j, i));
		}
	}
}

int realMain(int argc, char** argv)
//int main(int argc, char** argv)
{
	string fname = argv[1];
	string file_name = ".\\data\\" + fname;
	int n = atoi(argv[2]);
	FILE* fp;
#ifdef _WIN64
	fopen_s(&fp, file_name.c_str(), "r");
#else
	fp = fopen(file_name.c_str(), "r");
#endif
	float* x = new float[n];
	float* y = new float[n];

	bool isOpen = getData(fp, x, y, n);
	if (!isOpen) {
		return -1;
	}

	string name[2] = { "Single Method", "Multi Method" };
	DS_timer timer(2);
	for (int i = 0; i < 2; i++)
		timer.setTimerName(i, name[i]);

	float* resultAffinSingle = new float[n * n];
	generateAffinityMatrix(x, y, n, resultAffinSingle);
	float* serialResult = generateLaplacianMatrix(resultAffinSingle, n);

	MatrixXd A(n, n);
	for (int i = 0; i < n * n; i++) {
		A(i / n, i % n) = serialResult[i];
	}

	MatrixXd eigenVectors(n, n);
	VectorXd eigenValues(n);
	EigenSolver<MatrixXd> solver = EigenSolver<MatrixXd>(A);

	MatrixXcd solveEigenVector = solver.eigenvectors();
	VectorXcd solveEigenValue = solver.eigenvalues();

	for (int i = 0; i < n; i++) {
		eigenValues[i] = solveEigenValue[i].real();
		for (int j = 0; j < n; j++) {
			eigenVectors(i, j) = solveEigenVector(i, j).real();
		}
	}

	int* results = new int[n];
	find_2nd_Min(eigenValues, eigenVectors);
	labelDecomposition(eigenVectors, results, n);




	bool isCorrect = true;
	int idx[2] = { 0 };
	/*
	for (int i = 0; i < n; i++) {
		for (int j = 0; j < n; j++) {
			double single = dataSingle[i][j];
			if (abs(single - dataMuliti1[i][j]) > TOL || abs(single - dataMuliti2[i][j]) > TOL || abs(single - dataMuliti3[i][j]) > TOL
				|| abs(single - dataMuliti4_1[i][j]) > TOL || abs(single - dataMuliti4_2[i][j]) > TOL) {
				isCorrect = false;
				idx[0] = i; idx[1] = j;
				break;
			}
		}
	}
	*/
	/*
	for (int i = 0; i < n; i++) {
		for (int j = 0; j < n; j++) {
			double single = dataSingle[i][j];
			if (abs(single - dataMuliti1[i][j]) > TOL) {
				isCorrect = false;
				idx[0] = i; idx[1] = j;
				break;
			}
		}
	}
	*/
	if (isCorrect) {
		printf("Data is correct.\n");
	}
	else {
		printf("Data is not correct. ");
		//printf("Single[%d][%d] : %lf\n", idx[0], idx[1], dataSingle[idx[0]][idx[1]]);
		//printf("dataMuliti[%d][%d] : %lf\n", idx[0], idx[1], dataMuliti1[idx[0]][idx[1]]);
		/*
		printf("dataMuliti2[%d][%d] : %lf\n", idx[0], idx[1], dataMuliti2[idx[0]][idx[1]]);
		printf("dataMuliti3[%d][%d] : %lf\n", idx[0], idx[1], dataMuliti3[idx[0]][idx[1]]);
		printf("dataMuliti4_1[%d][%d] : %lf\n", idx[0], idx[1], dataMuliti4_1[idx[0]][idx[1]]);
		printf("dataMuliti4_2[%d][%d] : %lf\n", idx[0], idx[1], dataMuliti4_2[idx[0]][idx[1]]);
		*/
	}

	for (int i = 0; i < 8; i++)
	{
		file_name.pop_back();
	}

	string version[] = { "SingleMethod", "MultiMethod" };
	timer.printTimer();

	string saveName;
	saveName = file_name + "Serial_result.txt";
	saveData(saveName.c_str(), x, y, results, n);


	delete[] x, y, resultAffinSingle, results;

	return 0;
}